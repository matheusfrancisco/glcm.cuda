#include "hip/hip_runtime.h"
#include "DICOMReader.h"
#include "features.h"
#include "file.h"
#include "glcm_gpu.h"
#include "image.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <unordered_map>

using namespace std;
namespace fs = std::filesystem;

void checkCudaError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error after %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void apply_glcm_1(int *matrix, int max, int n_row, int n_col,
                  std::string result_csv, std::string filename = "default",
                  bool write_output = false) {

  int dx_array[] = {1, 1, 0, -1, -1, -1, 0, -1};
  int dy_array[] = {0, -1, -1, -1, 0, 1, 1, 1};
  int num_directions = 8;

  int glcm_size = (max * max) * sizeof(int);

  int *d_matrix;

  int *d_glcm;
  // Define CUDA kernel launch parameters
  int threads_per_block = 256;
  int total_pairs = n_row * (n_col - 1);
  int number_of_blocks =
      (total_pairs + threads_per_block - 1) / threads_per_block;

  hipMalloc((void **)&d_matrix, sizeof(int) * n_row * n_col);
  // Copy matrix to device
  hipMemcpy(d_matrix, matrix, sizeof(int) * n_row * n_col,
             hipMemcpyHostToDevice);

  std::unordered_map<std::string, double> time_map;
  std::unordered_map<std::string, double> total_gpu;
  auto start_time_global = std::chrono::high_resolution_clock::now();

  std::vector<float *> h_glcm_cuda_vec(num_directions, nullptr);
  // std::vector<int *> h_glcm_cuda_vec(num_directions, nullptr);

  for (int dir = 0; dir < num_directions; dir++) {

    int dx = dx_array[dir];
    int dy = dy_array[dir];
    std::cout << "Direction: " << dir << " dx: " << dx << " dy: " << dy
              << std::endl;

    // std::cout << "CudaMalloc: " << dir << std::endl;
    hipMalloc(&d_glcm, glcm_size);
    checkCudaError("hipMalloc d_glcm");
    hipMemset(d_glcm, 0, glcm_size);

    auto start_time = std::chrono::high_resolution_clock::now();
    checkCudaError("hipMemset d_glcm");
    glcm_cuda_direction<<<number_of_blocks, threads_per_block>>>(
        d_matrix, d_glcm, n_col, n_row, max, dx, dy);
    checkCudaError("glcm_cuda_optimized kernel launch");

    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
    checkCudaError("hipDeviceSynchronize");
    // Copy GLCM back to host
    int *h_glcm_cuda = (int *)malloc((max * max) * sizeof(int));

    hipMemcpy(h_glcm_cuda, d_glcm, sizeof(int) * (max * max),
               hipMemcpyDeviceToHost);

    checkCudaError("hipMemcpy to h_glcm_cuda");

    // needs to normalize the glcm matrix
    int sum = 0;
    for (int i = 0; i < max * max; i++) {
      sum += h_glcm_cuda[i];
    }

    int *d_g_glcm;
    hipMalloc((void **)&d_g_glcm, sizeof(int) * n_row * n_col);
    // Copy matrix to device
    hipMemcpy(d_g_glcm, h_glcm_cuda, sizeof(int) * n_row * n_col,
               hipMemcpyHostToDevice);

    checkCudaError("move glcm from thos to device");

    float *h_glcm_cuda_normalized;
    hipMalloc(&h_glcm_cuda_normalized,
               ((max + 1) * (max + 1)) * sizeof(float));
    checkCudaError("malloc glcm for cuda normalized");
    hipMemset(h_glcm_cuda_normalized, 0.f,
               ((max + 1) * (max + 1)) * sizeof(float));
    checkCudaError("set normalized");

    norm<<<256, 256>>>(d_g_glcm, h_glcm_cuda_normalized, max, sum);

    hipDeviceSynchronize();
    float *normalized =
        (float *)malloc(((max + 1) * (max + 1)) * sizeof(float));

    hipMemcpy(normalized, h_glcm_cuda_normalized,
               sizeof(float) * ((max + 1) * (max + 1)), hipMemcpyDeviceToHost);

    checkCudaError("Copy normalized glcm");

    h_glcm_cuda_vec[dir] = normalized;
    // extracting features from normalized glcm matrix
    float *contrast_value;
    hipMallocManaged(&contrast_value, sizeof(float) * (max * max));
    checkCudaError("Initialize contrast");
    // copy normalized matrix to device again

    float *normalized_glcm;
    hipMalloc((void **)&normalized_glcm,
               ((max + 1) * (max + 1)) * sizeof(float));
    checkCudaError("malloc normalized");

    // Copy matrix to device
    hipMemcpy(normalized_glcm, normalized,
               sizeof(float) * ((max + 1) * (max + 1)), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize(max + blockSize.x - 1 / blockSize.x,
                  (max + blockSize.y - 1) / blockSize.y);

    contrast<<<gridSize, blockSize>>>(normalized_glcm, contrast_value, max);

    hipDeviceSynchronize();
    hipMemcpy(contrast_value, contrast_value, sizeof(float),
               hipMemcpyDeviceToHost);

    std::cout << "Contrast: " << contrast_value[0] << std::endl;

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;

    time_map[filename + "_" + std::to_string(dir)] = elapsed.count();
    std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";

    hipFree(normalized_glcm);
    hipFree(d_glcm);
  }

  auto end_time_global = std::chrono::high_resolution_clock::now();
  total_gpu["total_gpu_dcm"] =
      std::chrono::duration<double>(end_time_global - start_time_global)
          .count();

  if (write_output) {
    for (int dir = 0; dir < num_directions; dir++) {
      std::cout << "dir: " << dir << std::endl;
      std::string r;
      {
        std::cout << "Writing output: " << filename.c_str() << std::endl;
        std::string path = filename;
        std::size_t last_slash = path.find_last_of("/\\");
        std::size_t second_last_slash =
            path.find_last_of("/\\", last_slash - 1);
        std::string file_path = path.substr(last_slash + 1);

        std::string part1 = path.substr(second_last_slash + 1,
                                        last_slash - second_last_slash - 1);
        std::string part2 = path.substr(last_slash + 1, path.find_last_of('.') -
                                                            last_slash - 1);

        std::string new_file_name =
            "/home/chico/m/chico/glcm.cuda/data/result/" + part1 + "-" + part2 +
            "_" + std::to_string(dir) + "_gpu_result.txt";
        std::cout << "Writing output: " << new_file_name << std::endl;
        r = new_file_name.c_str();
      }

      // write_image_matrix_glcm(r, h_glcm_cuda_vec[dir], max, max);
      write_image_matrix(r, h_glcm_cuda_vec[dir], max + 1, max + 1);
      free(h_glcm_cuda_vec[dir]);
    }
  }

  write_map_to_csv(time_map, result_csv);
  write_map_to_csv(total_gpu, "../total_gpu.csv");

  hipFree(d_matrix);
}

int main() {
  std::string folder = "/home/chico/m/chico/glcm.cuda/data";

  std::string folder_dcm = "/home/chico/m/chico/glcm.cuda/dataset";

  std::unordered_map<fs::path, fs::path, PathHash> file_map =
      get_images(folder);

  std::unordered_map<fs::path, fs::path, PathHash> file_map2 =
      get_images(folder_dcm);

  int test_flag = 1;

  if (test_flag == 1) {

    // auto file = file_map2.begin();

    string file =
        "/home/chico/m/chico/glcm.cuda/dataset/ST000001/SE000007/IM0000033.dcm";
    DICOMImage image;

    // std::cout << "Reading DICOM file: " << file.first.string() <<
    // std::endl;
    if (readDICOMImage(file, image)) {
      std::cout << "Image Dimensions: " << image.rows << " x " << image.cols
                << std::endl;

      // Example: Accessing pixel data
      if (!image.pixelData.empty()) {
        std::cout << "First pixel intensity: " << image.pixelData[0]
                  << std::endl;
      }

      int *matrix = (int *)malloc((image.rows * image.cols) * sizeof(int));
      int max = 0;
      for (int i = 0; i < image.rows * image.cols; i++) {
        matrix[i] = image.pixelData[i];
        if (image.pixelData[i] > max) {
          max = image.pixelData[i];
        }
      }

      if (max < 10000) {
        std::string r =
            "../data/csv_result/dcm_result" + std::to_string(0) + ".csv";
        apply_glcm_1(matrix, max, image.rows, image.cols, r, file, true);
      }

    } else {
      std::cerr << "Failed to read DICOM file." << std::endl;
    }
  }

  else {
    for (const auto &file : file_map) {
      std::string f = file.first.string();
      std::cout << f.c_str() << std::endl;

      png_image image_png;
      std::cout << f.c_str() << std::endl;

      // open the image png and put it into an array
      open_image_value_32b_array(f.c_str(), &image_png);

      size_t m_size = (image_png.width * image_png.height) * sizeof(int);
      int *matrix = (int *)malloc(m_size);
      // get the maximum valur of the image
      int max = 0;
      for (int i = 0; i < (image_png.height * image_png.width); ++i) {
        matrix[i] = image_png.image[i];
        if (matrix[i] > max) {
          max = matrix[i];
        }
      }
      max += 2;
      apply_glcm_1(matrix, max, image_png.height, image_png.width,
                   "../data/csv_result/png_result.csv", f, true);
      std::cout << "done" << std::endl;
    }

    int count = 0;
    for (const auto &file : file_map2) {
      DICOMImage image;

      // std::cout << "Reading DICOM file: " << file.first.string() <<
      // std::endl;
      if (readDICOMImage(file.first.string(), image)) {
        std::cout << "Image Dimensions: " << image.rows << " x " << image.cols
                  << std::endl;

        // Example: Accessing pixel data
        if (!image.pixelData.empty()) {
          std::cout << "First pixel intensity: " << image.pixelData[0]
                    << std::endl;
        }

        int *matrix = (int *)malloc((image.rows * image.cols) * sizeof(int));
        int max = 0;
        for (int i = 0; i < image.rows * image.cols; i++) {
          matrix[i] = image.pixelData[i];
          if (image.pixelData[i] > max) {
            max = image.pixelData[i];
          }
        }
        if (max < 10000) {
          std::string r =
              "../data/csv_result/dcm_result" + std::to_string(count) + ".csv";
          apply_glcm_1(matrix, max, image.rows, image.cols, r,
                       file.first.string(), true);
        }

      } else {
        std::cerr << "Failed to read DICOM file." << std::endl;
        continue;
      }
      count++;
    }
  }
  hipDeviceSynchronize();

  return 0;
}
