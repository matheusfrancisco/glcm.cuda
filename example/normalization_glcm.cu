#include "hip/hip_runtime.h"
#include "../DICOMReader.h"
#include "../file.h"
#include "../glcm_gpu.h"
#include "../image.h"
#include "omp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <unordered_map>

using namespace std;
namespace fs = std::filesystem;

void checkCudaError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error after %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void apply_glcm_1(int *matrix, int max, int n_row, int n_col,
                  std::string result_csv, std::string filename = "default",
                  bool write_output = false) {

  // std::cout << filename << std::endl;
  int dx_array[] = {0, 1, 1, 1, 0, -1, -1, -1};
  int dy_array[] = {1, 1, 0, -1, -1, -1, 0, 1};
  int num_directions = 8;

  int *d_matrix;

  // Define CUDA kernel launch parameters

  hipMalloc((void **)&d_matrix, sizeof(int) * n_row * n_col);
  // Copy matrix to device
  hipMemcpy(d_matrix, matrix, sizeof(int) * n_row * n_col,
             hipMemcpyHostToDevice);

  std::vector<int *> h_glcm_cuda_vec(num_directions, nullptr);
  std::vector<float *> h_glcm_cuda_vec2(num_directions, nullptr);

  std::vector<string> degree(8);
  int glcm_size = (max * max) * sizeof(int);

  int num_threads = 8;
  omp_set_num_threads(num_threads);
#pragma omp parallel for
  for (int dir = 0; dir < num_directions; dir++) {

    int *d_glcm;
    int dx = dx_array[dir];
    int dy = dy_array[dir];
    // std::cout << "Direction: " << dir << " dx: " << dx << " dy: " << dy
    //           << std::endl;

    // std::cout << "CudaMalloc: " << dir << std::endl;
    hipMalloc(&d_glcm, glcm_size);
    checkCudaError("hipMalloc d_glcm");
    hipMemset(d_glcm, 0, glcm_size);

    checkCudaError("hipMemset d_glcm");

    int threads_per_block = 256;
    int total_pairs = n_row * (n_col - 1);
    int number_of_blocks =
        (total_pairs + threads_per_block - 1) / threads_per_block;
    if (dx == 0 and dy == 1) { // 0
      glcm_cuda_direction<<<n_col, n_row>>>(d_matrix, d_glcm, n_col, n_row, max,
                                            dy, dx);

      degree[0] = "0";
    }

    else if (dx == 1 and dy == 1) { // 45 degree
      glcm_cuda_direction<<<number_of_blocks, threads_per_block>>>(
          d_matrix, d_glcm, n_col, n_row, max, dy, dx);

      degree[1] = "45";

    } else if (dx == 1 and dy == 0) // 90
    {
      glcm_cuda_direction<<<number_of_blocks, threads_per_block>>>(
          d_matrix, d_glcm, n_col, n_row, max, dy, dx);
      degree[2] = "90";
    }

    else if (dx == 1 and dy == -1) // 135 degree
    {
      glcm_cuda_direction<<<n_col, n_row>>>(d_matrix, d_glcm, n_col, n_row, max,
                                            dy, dx);

      degree[3] = "135";
    } else if (dx == 0 and dy == -1) { //  180

      glcm_cuda_direction<<<number_of_blocks, threads_per_block>>>(
          d_matrix, d_glcm, n_col, n_row, max, dy, dx);

      degree[4] = "180";
    }

    else if (dx == -1 and dy == -1) // 225
    {
      glcm_cuda_direction<<<n_col, n_row>>>(d_matrix, d_glcm, n_col, n_row, max,
                                            dy, dx);
      degree[5] = "225";
    }

    else if (dx == -1 and dy == 0) { // 270 degree

      glcm_cuda_direction<<<number_of_blocks, threads_per_block>>>(
          d_matrix, d_glcm, n_col, n_row, max, dy, dx);

      degree[6] = "270";
    } else if (dx == -1 and dy == 1) // 315
    {
      glcm_cuda_direction<<<n_col, n_row>>>(d_matrix, d_glcm, n_col, n_row, max,
                                            dy, dx);

      degree[7] = "315";
    }

    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
    checkCudaError("hipDeviceSynchronize");
    // Copy GLCM back to host
    int *h_glcm_cuda = (int *)malloc((max * max) * sizeof(int));
    hipMemcpy(h_glcm_cuda, d_glcm, sizeof(int) * (max * max),
               hipMemcpyDeviceToHost);

    checkCudaError("hipMemcpy to h_glcm_cuda");
    int enabled_normalization = 0;

    if (enabled_normalization == 1) {
      // #TODO normaization the glcm for be more easy to calculate some features

      // needs to normalize the glcm matrix
      int sum = 0;
      for (int i = 0; i < max * max; i++) {
        sum += h_glcm_cuda[i];
      }
      // std::cout << "sum elements: " << sum << std::endl;

      int *d_g_glcm;
      hipMalloc((void **)&d_g_glcm, sizeof(int) * max * max);
      //  Copy matrix to device
      hipMemcpy(d_g_glcm, h_glcm_cuda, sizeof(int) * max * max,
                 hipMemcpyHostToDevice);

      checkCudaError("move glcm from thos to device");

      float *h_glcm_cuda_normalized;
      hipMallocManaged(&h_glcm_cuda_normalized, max * max * sizeof(float));
      checkCudaError("malloc glcm for cuda normalized");

      hipMemset(h_glcm_cuda_normalized, 0.f, max * max * sizeof(float));
      checkCudaError("set normalized");

      norm<<<256, 256>>>(d_g_glcm, h_glcm_cuda_normalized, max, sum);

      hipDeviceSynchronize();
      float *normalized = (float *)malloc(max * max * sizeof(float));

      hipMemcpy(normalized, h_glcm_cuda_normalized, sizeof(float) * max * max,
                 hipMemcpyDeviceToHost);

      checkCudaError("Copy normalized glcm");

      // copy normalized matrix to device again

      float *normalized_glcm;
      hipMallocManaged((void **)&normalized_glcm, (max * max) * sizeof(float));
      checkCudaError("malloc normalized");

      // Copy matrix to device
      hipMemcpy(normalized_glcm, normalized, sizeof(float) * (max * max),
                 hipMemcpyHostToDevice);

      // extracting features from normalized glcm matrix
      // extracting contrast
      // float *contrast_value;
      // hipMallocManaged(&contrast_value, (max * max) * sizeof(float));
      // checkCudaError("Initialize contrast");

      //
      // dim3 blockSize(32, 32);
      // dim3 gridSize((max + blockSize.x - 1) / blockSize.x, (max +
      // blockSize.y
      // - 1) / blockSize.y);
      //// features
      // contrast<<<gridSize, blockSize>>>(normalized_glcm, contrast_value,
      // max); hipDeviceSynchronize(); printf("Contrast: %.4f\n",
      // contrast_value[0]);
      h_glcm_cuda_vec2[dir] = normalized_glcm;
    } else {

      h_glcm_cuda_vec[dir] = h_glcm_cuda;
    }

    hipFree(d_glcm);

    if (write_output) {

      // std::cout << "dir: " << dir << std::endl;
      // std::cout << "Writing output dir: " << degree[dir] << std::endl;
      std::string r;
      {
        // std::cout << "Writing output: " << filename.c_str() << std::endl;
        std::string path = filename;
        std::size_t last_slash = path.find_last_of("/\\");
        std::size_t second_last_slash =
            path.find_last_of("/\\", last_slash - 1);
        std::string file_path = path.substr(last_slash + 1);

        std::string part1 = path.substr(second_last_slash + 1,
                                        last_slash - second_last_slash - 1);
        std::string part2 = path.substr(last_slash + 1, path.find_last_of('.') -
                                                            last_slash - 1);

        std::string new_file_name =
            "/home/chico/m/chico/glcm.cuda/data/result/" + part1 + "-" + part2 +
            "_" + std::to_string(dir) + "_" + degree[dir] + "_gpu_result.txt";
        // std::cout << "Writing output: " << new_file_name << std::endl;
        r = new_file_name.c_str();
      }

      write_image_matrix_glcm(r, h_glcm_cuda_vec[dir], max, max);
      //      int enabled_normalization = 0;
      //      if (enabled_normalization == 1) {
      //        write_image_matrix(r, h_glcm_cuda_vec2[dir], max, max);
      //      }
      free(h_glcm_cuda_vec[dir]);
    }
  }

  hipFree(d_matrix);
}

int main() {
  std::string folder = "/home/chico/m/chico/glcm.cuda/data/images_png/";

  std::string folder_dcm = "/home/chico/m/chico/glcm.cuda/dataset";

  std::unordered_map<fs::path, fs::path, PathHash> file_map =
      get_images(folder);

  std::unordered_map<fs::path, fs::path, PathHash> file_map2 =
      get_images(folder_dcm);

  std::vector<std::filesystem::path> file_map4 = {
      /* populate with file paths */};
  for (const auto &entry : file_map) {
    file_map4.push_back(entry.first);
  }
  std::unordered_map<std::string, double> total_gpu_jpg;
  auto start_time_global_jpg = std::chrono::high_resolution_clock::now();
  int num_threads = 355;
  omp_set_num_threads(num_threads);
  // #For images jpg
// to run without prgrama command the line bellow
#pragma omp parallel for
  for (size_t i = 0; i < file_map4.size(); ++i) {
    const auto &file = file_map4[i];
    std::string f = file.string();

    png_image image_png;

    // open the image png and put it into an array
    open_image_value_32b_array(f.c_str(), &image_png);

    size_t m_size = (image_png.width * image_png.height) * sizeof(int);
    int *matrix = (int *)malloc(m_size);
    // get the maximum valur of the image
    int max = 0;
    for (int i = 0; i < (image_png.height * image_png.width); ++i) {
      matrix[i] = image_png.image[i];
      if (matrix[i] > max) {
        max = matrix[i];
      }
    }
    max += 2;
    apply_glcm_1(matrix, max, image_png.height, image_png.width,
                 "../data/csv_result/png_result.csv", f, true);
    std::cout << "Reading PNG file: " << i << std::endl;
  }

  auto end_time_global_jpg = std::chrono::high_resolution_clock::now();
  total_gpu_jpg["total_gpu_jpg"] =
      std::chrono::duration<double>(end_time_global_jpg - start_time_global_jpg)
          .count();

  write_map_to_csv(total_gpu_jpg, "../total_gpu_jpg.csv");

  std::vector<std::filesystem::path> file_map3 = {
      /* populate with file paths */};
  for (const auto &entry : file_map2) {
    file_map3.push_back(entry.first);
  }

  std::unordered_map<std::string, double> total_gpu;
  auto start_time_global = std::chrono::high_resolution_clock::now();
#pragma omp parallel for
  for (size_t i = 0; i < file_map3.size(); ++i) {
    const auto &file = file_map3[i];
    std::cout << "Reading DICOM file: " << i << std::endl;
    DICOMImage image;

    // std::cout << "Reading DICOM file: " << file.first.string() <<
    // std::endl;
    if (readDICOMImage(file.string(), image)) {

      int *matrix = (int *)malloc((image.rows * image.cols) * sizeof(int));
      int max = 0;
      for (int i = 0; i < image.rows * image.cols; i++) {
        matrix[i] = image.pixelData[i];
        if (image.pixelData[i] > max) {
          max = image.pixelData[i];
        }
      }
      max += 1;

      if (max < 10000) {
        std::string r =
            "../data/csv_result/dcm_result" + std::to_string(i) + ".csv";
        apply_glcm_1(matrix, max, image.rows, image.cols, r, file.string(),
                     true);
      }
      free(matrix);
    } else {
      // std::cerr << "Failed to read DICOM file." << std::endl;
      continue;
    }
  }

  auto end_time_global = std::chrono::high_resolution_clock::now();
  total_gpu["total_gpu_dcm"] =
      std::chrono::duration<double>(end_time_global - start_time_global)
          .count();

  write_map_to_csv(total_gpu, "../total_gpu.csv");

  return 0;
}
